#include "hip/hip_runtime.h"
#include <stdio.h>
#include "timer.h"

struct Matrix {
  int channels;
  int width;
  int height;
  int stride;
  double* elements;
}; 

struct Filter {
    int fh;
    int fw;
    int c;
    int k;
    double* weights;
};

Matrix MakeDeviceMatrix(Matrix M, bool copy){
  // Create a new matrix in device memory.
  Matrix newDeviceMatrix;
  newDeviceMatrix.width = M.width;
  newDeviceMatrix.stride = M.width;
  newDeviceMatrix.height = M.height;
  newDeviceMatrix.channels = M.channels;
  size_t size = M.channels * M.width * M.height * sizeof(double);
  hipMalloc((void**) &newDeviceMatrix.elements, size);
  if (copy)
    hipMemcpy(newDeviceMatrix.elements, M.elements, size, hipMemcpyHostToDevice);
  return newDeviceMatrix;
}

Filter MakeDeviceFilter(Filter F, bool copy){
    // Create a new filter in device memory.
    Filter newDeviceFilter;
    newDeviceFilter.fh = F.fh;
    newDeviceFilter.fw = F.fw;
    newDeviceFilter.c = F.c;
    newDeviceFilter.k = F.k;
    size_t size = F.fh * F.fw * F.c * F.k * sizeof(double);
    hipMalloc((void**) &newDeviceFilter.weights, size);
    if (copy)
        hipMemcpy(newDeviceFilter.weights, F.weights, size, hipMemcpyHostToDevice);
    return newDeviceFilter;
}

Matrix MakeHostMatrix(int channels, int width, int height, bool use_zeros){
  //row major 
  Matrix newHostMatrix;
  newHostMatrix.width = width;
  newHostMatrix.stride = width;
  newHostMatrix.height = height;
  newHostMatrix.channels = channels;
  size_t size = channels * width * height * sizeof(double);
  newHostMatrix.elements = (double*)malloc(size);
  int grid_size = width * height;
  int nrows = height;
  int ncols = width;
  for(int c=0; c < channels; c++){
    for (int x = 0; x < nrows; x++){
        for (int y=0; y < ncols; y++){
            if (use_zeros){
                newHostMatrix.elements[c*grid_size + x * width + y] = 0;
            }else{
                double elem = c * (x + y);
                newHostMatrix.elements[c*grid_size + x * width + y] = elem;
            }
            
        }
    }
  }
  return newHostMatrix;
}

Matrix PadMatrix(Matrix M, int padding){
    Matrix newMatrix;
    newMatrix.width = M.width + 2 * padding;
    newMatrix.stride = M.width + 2 * padding;
    newMatrix.height = M.height + 2 * padding;
    newMatrix.channels = M.channels;
    size_t size = M.channels * newMatrix.width * newMatrix.height * sizeof(double);
    newMatrix.elements = (double*)malloc(size);
    int new_grid_size = newMatrix.width * newMatrix.height;
    int old_grid_size = M.width * M.height;
    for(int c=0; c < M.channels; c++){
        for (int x=0; x < newMatrix.height; x++){
            for (int y=0; y < newMatrix.width; y++){
               /*
               for case 3, 1024, 1024, this gets triggered when x/y = 0 or 
               */
               if(x < padding || x >= M.height+padding || y < padding || y >= M.width+padding){
                   newMatrix.elements[c * new_grid_size + x * newMatrix.width + y] = 0;
               }
               else{
                   int orig_x = x - padding;
                   int orig_y = y - padding;
                //    if (x == 1025){
                //     printf("should not be here");
                //    }
                   double t_elem = M.elements[c * old_grid_size + orig_x * M.width + orig_y];
                   newMatrix.elements[c * new_grid_size + x * newMatrix.width + y] = t_elem;
                   
               }
            }
        }
    }
    return newMatrix;
}

Filter MakeHostFilter(int c_in, int k_out, int fw, int fh){
    //row major
    Filter newHostFilter;
    newHostFilter.c = c_in;
    newHostFilter.k = k_out;
    newHostFilter.fw = fw;
    newHostFilter.fh = fh;
    size_t size = c_in * k_out * fw * fh * sizeof(double);
    newHostFilter.weights = (double*)malloc(size);
    int grid_size = fw * fh;
    for(int k=0; k < k_out; k++){
        for (int c=0; c < c_in; c++){
            for (int i=0; i < fh; i++){
                for (int j=0; j < fw; j++){
                    double elem = (c + k) * (i + j);
                    //k * c_in * grid_size takes you to the correct out filter 
                    // c * grid_size takes you to the right channel 
                    // j *fw takes you to the right row 
                    // i takes you to the right column
                    newHostFilter.weights[k * c_in * grid_size + c * grid_size + i * fw + j] = elem;
                }
            }
        }
    }
    return newHostFilter;
}

// void PrintChannel(Matrix M, int channel ){
//     int grid_size = M.width * M.height;
//     for(int x= 0; x < M.height; x++){
//         for(int y=0; y < M.width; y++){
//             printf("%.0f,", M.elements[grid_size* channel + x * M.width + y]);
//         }
//         printf("\n");
//     }
// }

// void PrintFilter(Filter F, int k ){
//     int grid_size = F.fw * F.fh;
//     for (int c=0; c < F.c; c++){
//         printf("FChannel: %d\n", c);
//         for (int i=0; i < F.fh; i++){
//             for (int j=0; j < F.fw; j++){
//                 printf("%.0f,", F.weights[k * F.c * grid_size + c * grid_size + i * F.fw + j]);
//             }
//             printf("\n");
//         }
//     }
// }

__global__ void NaiveConvKernel(Matrix M, Filter F, Matrix O){
    /*
    keep it simple: each thread is responsible for one element in the output, 
    so we use a single 1024 x 1 thread block, and then a 64 x 1024 grid of blocks.
    
    */

    int row_idx = blockIdx.y; 
    int col_idx = threadIdx.x;
    int c_out_idx = blockIdx.x;
    int elem_per_filter = F.fw * F.fh * F.c;

    //if(c_out_idx == 0 && row_idx == 0 && col_idx == 0) PrintFilter(F, 0)
    
    double* Msub;
    double* Fsub;
    Msub = &M.elements[row_idx * M.width + col_idx];// only need to worry about getting into the first channel 
    Fsub = &F.weights[c_out_idx * elem_per_filter ]; //only need to get to start of right out channel
    
    double sum = 0;
    for (int c = 0; c < 3 ; c++){
        for (int x = 0; x < 3; x++){
            for (int y = 0; y < 3; y++){
                sum += Msub[c * (1026*1026) + x * 1026 + y] * Fsub[c * (3*3) + x * 3 + y];
            }
        }
    }
    // if (c_out_idx == 0 && row_idx == 0 && col_idx == 0){
    //     printf("sum: %f\n", sum);
    // };

    O.elements[c_out_idx * (1024*1024) + row_idx * 1024 + col_idx] = sum;
}



int main(){
    // borrowed timing code from Part A
    Matrix h_mat = MakeHostMatrix(3, 1024, 1024, false);
    Matrix h_padded_mat = PadMatrix(h_mat, 1);
    Matrix h_out_mat = MakeHostMatrix(64, 1024, 1024, true);
    Filter h_filt = MakeHostFilter(3, 64, 3, 3);
    
    //PrintChannel(h_mat, 2);

    Matrix d_mat = MakeDeviceMatrix(h_padded_mat, true);
    Filter d_filt = MakeDeviceFilter(h_filt, true);
    Matrix d_out_mat = MakeDeviceMatrix(h_out_mat, false);
    dim3 dimBlock(1024);
    dim3 dimGrid(64,1024);
    // warmup
    NaiveConvKernel<<<dimGrid, dimBlock>>>(d_mat, d_filt, d_out_mat);
    hipDeviceSynchronize();

    // run for real 
    initialize_timer();
    start_timer();
    NaiveConvKernel<<<dimGrid, dimBlock>>>(d_mat, d_filt, d_out_mat);
    hipDeviceSynchronize() ;
    stop_timer();
    double time = elapsed_time();
    hipMemcpy(h_out_mat.elements, d_out_mat.elements, 64 * 1024 * 1024 * sizeof(double), hipMemcpyDeviceToHost);
    double checksum = 0;
    for(int i = 0; i < 64 * 1024 * 1024; i++){
        checksum += h_out_mat.elements[i];
    }
    printf("%.1f, %.3f\n", checksum, time*1000);
    //PrintChannel(h_out_mat, 0);
    return 0 ;
}
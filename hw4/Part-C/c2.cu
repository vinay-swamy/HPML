#include "hip/hip_runtime.h"
#include <stdio.h>
#include "timer.h"

struct Matrix {
  int channels;
  int width;
  int height;
  int stride;
  double* elements;
}; 

struct Filter {
    int fh;
    int fw;
    int c;
    int k;
    double* weights;
};

Matrix MakeDeviceMatrix(Matrix M, bool copy){
  // Create a new matrix in device memory.
  Matrix newDeviceMatrix;
  newDeviceMatrix.width = M.width;
  newDeviceMatrix.stride = M.width;
  newDeviceMatrix.height = M.height;
  newDeviceMatrix.channels = M.channels;
  size_t size = M.channels * M.width * M.height * sizeof(double);
  hipMalloc((void**) &newDeviceMatrix.elements, size);
  if (copy)
    hipMemcpy(newDeviceMatrix.elements, M.elements, size, hipMemcpyHostToDevice);
  return newDeviceMatrix;
}

Filter MakeDeviceFilter(Filter F, bool copy){
    // Create a new filter in device memory.
    Filter newDeviceFilter;
    newDeviceFilter.fh = F.fh;
    newDeviceFilter.fw = F.fw;
    newDeviceFilter.c = F.c;
    newDeviceFilter.k = F.k;
    size_t size = F.fh * F.fw * F.c * F.k * sizeof(double);
    hipMalloc((void**) &newDeviceFilter.weights, size);
    if (copy)
        hipMemcpy(newDeviceFilter.weights, F.weights, size, hipMemcpyHostToDevice);
    return newDeviceFilter;
}

Matrix MakeHostMatrix(int channels, int width, int height, bool use_zeros){
  //row major 
  Matrix newHostMatrix;
  newHostMatrix.width = width;
  newHostMatrix.stride = width;
  newHostMatrix.height = height;
  newHostMatrix.channels = channels;
  size_t size = channels * width * height * sizeof(double);
  newHostMatrix.elements = (double*)malloc(size);
  int grid_size = width * height;
  int nrows = height;
  int ncols = width;
  for(int c=0; c < channels; c++){
    for (int x = 0; x < nrows; x++){
        for (int y=0; y < ncols; y++){
            if (use_zeros){
                newHostMatrix.elements[c*grid_size + x * width + y] = 0;
            }else{
                double elem = c * (x + y);
                newHostMatrix.elements[c*grid_size + x * width + y] = elem;
            }
            
        }
    }
  }
  return newHostMatrix;
}

Matrix PadMatrix(Matrix M, int padding){
    Matrix newMatrix;
    newMatrix.width = M.width + 2 * padding;
    newMatrix.stride = M.width + 2 * padding;
    newMatrix.height = M.height + 2 * padding;
    newMatrix.channels = M.channels;
    size_t size = M.channels * newMatrix.width * newMatrix.height * sizeof(double);
    newMatrix.elements = (double*)malloc(size);
    int new_grid_size = newMatrix.width * newMatrix.height;
    int old_grid_size = M.width * M.height;
    for(int c=0; c < M.channels; c++){
        for (int x=0; x < newMatrix.height; x++){
            for (int y=0; y < newMatrix.width; y++){
               /*
               for case 3, 1024, 1024, this gets triggered when x/y = 0 or 
               */
               if(x < padding || x >= M.height+padding || y < padding || y >= M.width+padding){
                   newMatrix.elements[c * new_grid_size + x * newMatrix.width + y] = 0;
               }
               else{
                   int orig_x = x - padding;
                   int orig_y = y - padding;
                //    if (x == 1025){
                //     printf("should not be here");
                //    }
                   double t_elem = M.elements[c * old_grid_size + orig_x * M.width + orig_y];
                   newMatrix.elements[c * new_grid_size + x * newMatrix.width + y] = t_elem;
                   
               }
            }
        }
    }
    return newMatrix;
}

Filter MakeHostFilter(int c_in, int k_out, int fw, int fh){
    //row major
    Filter newHostFilter;
    newHostFilter.c = c_in;
    newHostFilter.k = k_out;
    newHostFilter.fw = fw;
    newHostFilter.fh = fh;
    size_t size = c_in * k_out * fw * fh * sizeof(double);
    newHostFilter.weights = (double*)malloc(size);
    int grid_size = fw * fh;
    for(int k=0; k < k_out; k++){
        for (int c=0; c < c_in; c++){
            for (int i=0; i < fh; i++){
                for (int j=0; j < fw; j++){
                    double elem = (c + k) * (i + j);
                    //k * c_in * grid_size takes you to the correct out filter 
                    // c * grid_size takes you to the right channel 
                    // j *fw takes you to the right row 
                    // i takes you to the right column
                    newHostFilter.weights[k * c_in * grid_size + c * grid_size + i * fw + j] = elem;
                }
            }
        }
    }
    return newHostFilter;
}

void PrintChannel(Matrix M, int channel ){
    int grid_size = M.width * M.height;
    for(int x= 0; x < M.height; x++){
        for(int y=0; y < M.width; y++){
            printf("%.0f,", M.elements[grid_size* channel + x * M.width + y]);
        }
        printf("\n");
    }
}

void PrintFilter(Filter F, int k ){
    int grid_size = F.fw * F.fh;
    for (int c=0; c < F.c; c++){
        printf("FChannel: %d\n", c);
        for (int i=0; i < F.fh; i++){
            for (int j=0; j < F.fw; j++){
                printf("%.0f,", F.weights[k * F.c * grid_size + c * grid_size + i * F.fw + j]);
            }
            printf("\n");
        }
    }
}

__global__ void TiledConvKernel(Matrix M, Filter F, Matrix O){
    
    int thread_row = threadIdx.y;
    int thread_col = threadIdx.x;
    int block_row = blockIdx.y;
    int block_col = blockIdx.x;
    int c_out_channel = blockIdx.z;
    int block_width = 30;
    int block_height = 30;
    int global_row = block_row * block_height+ thread_row;
    int global_col = block_col * block_width + thread_col;
    /*
    Final Solution - the main problem is that the the input and output are not the same size, 
    and we have to overlap the tiles by two to correctly get the output.
    Read in a 32x32 tile, using a 30x30 thread block. 1024/30 = 34.13 ~ 35, so we can have a 35 x35 grid,
    and then have 64 blocks in the Z direction, one for each output channel. So each convolutional filter operates
    on a 3x3x3 subregion of the tile in parallel, across the whole input matrix
    */
   
    double* Msub; 
    // because each block operates on a 3 x32 x32 tile, we only need to set the pointer 
    // to a value in the 0th channel; 
    Msub = &M.elements[block_row * (block_width * M.width) + block_col*block_width];
    // allocate shared as a flat array as well, to simply the indexing math a little bit
    __shared__ double shared_Msub[3*32*32];
    
    
    for (int c = 0; c < 3; c++){
        for( int os =0; os < 2; os++){
            // because we have the issue of the input and output not being the same size, we can 
            // think of this as operations along the flat array, and derive a total thread index, 
            // where each consecutuve thread reads consecutuve elements in the flat matrix
            // need to map this total thread index to a row and col in the global matrix 
            // each thread reads in 1 pixel, but across all channels.
            // additionally, bc we have 32x32 thread tile, but 30x30 threads, some threads will need
            // to read in 2 pixels, so we need to account for that as well, and make sure that we don't
            // read out of bounds
            int total_thread_idx = thread_row *30 + thread_col + os * (30*30);
            int row_in_Msub = total_thread_idx / 32;
            int col_in_Msub = total_thread_idx % 32;
            if (total_thread_idx < 32*32){
                shared_Msub[c*(32*32) + total_thread_idx] = Msub[c*(M.width*M.height) + row_in_Msub * M.width + col_in_Msub];
            }
        }
    }

    __shared__ double shared_Fsub[3*3*3];
    //each block reads in a single filter.
    // we have 27 elements in the filter, so can simply read the filter in as a flat array, 
    // and have each thread read in a single element, and have the rest of the threads chill
    int flat_thread_index = thread_row * 30 + thread_col;
    if (flat_thread_index < 3*3*3){
        shared_Fsub[flat_thread_index] = F.weights[c_out_channel * (3*3*3) + flat_thread_index];
    }

    __syncthreads();

    double sum = 0;
    for( int c = 0; c < 3 ;c++){
        for (int x = 0; x < 3; x++){
            for (int y = 0; y < 3 ; y++){
                //c * (3*3) + x * 3 +y
                //inchannel   row   col      
                if (global_row < 1024 && global_col < 1024){ //  only update if in bounds
                    double weight = shared_Fsub[c * (3*3) + x * 3 +y];
                    //c*(32*32) + (thread_row + x) * 32 + (thread_col + y)
                    //c*(32*32) right channel 
                    //(thread_row + x) - thread row takes us to the right row, and then x takes us to the right row in the filter operation
                    //(thread_col + y) - thread col takes us to the right col, and then y takes us to the right col in the filter operation
                    double elem = shared_Msub[c*(32*32) + (thread_row + x) * 32 + (thread_col + y)];
                    sum += weight * elem;
                }
            }
        }
    }
    
    //write to the output
    if (global_row < 1024 && global_col < 1024){
        O.elements[c_out_channel * (O.width * O.height) + global_row * 1024 + global_col] = sum;
    }
    
}

int main(int argc, char *argv[]){

    Matrix h_mat = MakeHostMatrix(3, 1024, 1024, false);
    Matrix h_padded_mat = PadMatrix(h_mat, 1);
    Matrix h_out_mat = MakeHostMatrix(64, 1024, 1024, true);
    Filter h_filt = MakeHostFilter(3, 64, 3, 3);
    
    //PrintChannel(h_padded_mat, 1);

    Matrix d_mat = MakeDeviceMatrix(h_padded_mat, true);
    Filter d_filt = MakeDeviceFilter(h_filt, true);
    Matrix d_out_mat = MakeDeviceMatrix(h_out_mat, false);
    
    // 
    dim3 dimBlock(30,30);
    //1024/30 = 34.13 ~ 35
    dim3 dimGrid(35,35,64);
    //warmup
    TiledConvKernel<<<dimGrid, dimBlock>>>(d_mat, d_filt, d_out_mat);
    hipDeviceSynchronize();
    // run for real 
    initialize_timer();
    start_timer();
    TiledConvKernel<<<dimGrid, dimBlock>>>(d_mat, d_filt, d_out_mat);
    hipDeviceSynchronize() ;
    stop_timer();
    double time = elapsed_time();
    hipMemcpy(h_out_mat.elements, d_out_mat.elements, 64 * 1024 * 1024 * sizeof(double), hipMemcpyDeviceToHost);
    double checksum = 0;
    for(int i = 0; i < 64 * 1024 * 1024; i++){
        checksum += h_out_mat.elements[i];
    }
    printf("%.1f, %.3f\n", checksum, time*1000);
    //int cout_channel = std::atoi(argv[1]);
    //PrintChannel(h_out_mat, cout_channel );
    return 0 ;
}